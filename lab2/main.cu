#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>


#define HANDLE_ERROR(err)                            \
    do { if (err != hipSuccess) { printf("ERROR: %s\n", hipGetErrorString(err)); exit(0);} } while (0)


texture<uchar4, 2, hipReadModeElementType> tex;


__global__ void kernel(uchar4 *dst, int w, int h, int wRatio, int hRatio)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int cntSamples = wRatio * hRatio;

    for (int x = idx; x < w; x += offsetx) {
        for (int y = idy; y < h; y += offsety) {
            int3 samples;
            samples.x = 0;
            samples.y = 0;
            samples.z = 0;
            for (int i = 0; i < wRatio; ++i) {
                for (int j = 0; j < hRatio; ++j) {
                    uchar4 p = tex2D(tex, x * wRatio + i, y * hRatio + j);
                    samples.x += p.x;
                    samples.y += p.y;
                    samples.z += p.z;
                }
            }
            samples.x /= cntSamples;
            samples.y /= cntSamples;
            samples.z /= cntSamples;
            dst[x + y * w] = make_uchar4(samples.x, samples.y, samples.z, 0);
        }
    }
}


int main()
{
    std::string input, output;
    int w, h, wNew, hNew;
    uchar4 *data;
    std::cin >> input >> output >> wNew >> hNew;

    std::ifstream fsIn(input, std::ios::in | std::ios::binary);
    if (fsIn.is_open()) {
        fsIn.read((char *)&w, sizeof(w));
        fsIn.read((char *)&h, sizeof(h));
        data = new uchar4[w * h];
        fsIn.read((char *)data, w * h * sizeof(data[0]));
        fsIn.close();
    } else {
        return 1;
    }

    int wRatio = w / wNew;
    int hRatio = h / hNew;

    std::cout << w << " " << h << std::endl;

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    HANDLE_ERROR(hipMallocArray(&arr, &ch, w, h));
    HANDLE_ERROR(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * h * w, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;

    HANDLE_ERROR(hipBindTextureToArray(tex, arr, ch));
    uchar4 *dev_data;
    HANDLE_ERROR(hipMalloc(&dev_data, sizeof(uchar4) * hNew * wNew));
    kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_data, wNew, hNew, wRatio, hRatio);
    HANDLE_ERROR(hipMemcpy(data, dev_data, sizeof(uchar4) * hNew * wNew, hipMemcpyDeviceToHost));

    std::ofstream fsOut(output, std::ios::out | std::ios::binary);
    if (fsOut.is_open()) {
        fsOut.write((char *)&wNew, sizeof(wNew));
        fsOut.write((char *)&hNew, sizeof(hNew));
        fsOut.write((char *)data, wNew * hNew * sizeof(data[0]));
        fsOut.close();
    } else {
        return 1;
    }

    HANDLE_ERROR(hipUnbindTexture(tex));
    HANDLE_ERROR(hipFreeArray(arr));
    HANDLE_ERROR(hipFree(dev_data));

    delete[] data;
    return 0;
}