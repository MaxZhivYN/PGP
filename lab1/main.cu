
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <algorithm>

using namespace std;

__global__ void minVector(double* arr1_gpu, double* arr2_gpu, double* res_gpu, int n) {
    for (int i = 0; i < n; ++i) {
        res_gpu[i] = min(arr1_gpu[i], arr2_gpu[i]);
    }
}

int main() {
    int n;
    cin >> n;
    
    double* arr1 = new double[n];
    double* arr2 = new double[n];
    double* res = new double[n];

    for (int i = 0; i < n; ++i) {
        cin >> arr1[i];
    }

    for (int i = 0; i < n; ++i) {
        cin >> arr2[i];
    }

    double* arr1_gpu;
    double* arr2_gpu;
    double* res_gpu;

    hipMalloc((void**) &arr1_gpu, sizeof(double) * n );
    hipMalloc((void**) &arr2_gpu, sizeof(double) * n);
    hipMalloc((void**) &res_gpu, sizeof(double) * n);

    hipMemcpy(arr1_gpu, arr1, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

    minVector<<<32, 32>>>(arr1_gpu, arr2_gpu, res_gpu, n);

    hipMemcpy(res, res_gpu, sizeof(double) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        cout << std::scientific << std::setprecision(10) << res[i] << " ";
    }

    cout << endl;

    hipFree(arr1_gpu);
    hipFree(arr2_gpu);
    hipFree(res_gpu);

    return 0;
}
