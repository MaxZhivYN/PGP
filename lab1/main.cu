
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <algorithm>

using namespace std;

__global__ void minVector(double* arr1_gpu, double* arr2_gpu, int n) {
    for (int i = 0; i < n; ++i) {
        arr1_gpu[i] = min(arr1_gpu[i], arr2_gpu[i]);
    }
}

int main() {
    int n;
    cin >> n;
    
    double* arr1 = new double[n];
    double* arr2 = new double[n];


    for (int i = 0; i < n; ++i) {
        cin >> arr1[i];
    }

    for (int i = 0; i < n; ++i) {
        cin >> arr2[i];
    }

    double* arr1_gpu;
    double* arr2_gpu;

    hipMalloc((void**) &arr1_gpu, sizeof(double) * n);
    hipMalloc((void**) &arr2_gpu, sizeof(double) * n);

    hipMemcpy(arr1_gpu, arr1, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

    minVector<<<256, 512>>>(arr1_gpu, arr2_gpu, n);

    hipMemcpy(arr1, arr1_gpu, sizeof(double) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        cout << std::scientific << std::setprecision(10) << arr1[i] << " ";
    }

    cout << endl;


    hipFree(arr1_gpu);
    hipFree(arr2_gpu);

    delete[] arr1;
    delete[] arr2;

    return 0;
}
