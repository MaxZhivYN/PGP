
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

using namespace std;

__global__ void minVector(double* arr1_gpu, double* arr2_gpu, double* res_gpu, int n) {
    for (int i = 0; i < n; ++i) {
        res_gpu[i] = min(arr1_gpu[i], arr2_gpu[i]);
    }
}

int main() {
    int n;
    cin >> n;
    
    double* arr1 = new double[n];
    double* arr2 = new double[n];
    double* res = new double[n];

    for (int i = 0; i < n; ++i) {
        cin >> arr1[i];
    }

    for (int i = 0; i < n; ++i) {
        cin >> arr2[i];
    }

    double* arr1_gpu;
    double* arr2_gpu;
    double* res_gpu;

    hipMalloc((void**) &arr1_gpu, sizeof(double) * n );
    hipMalloc((void**) &arr2_gpu, sizeof(double) * n);
    hipMalloc((void**) &res_gpu, sizeof(double) * n);

    hipMemcpy(arr1_gpu, arr1, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(arr2_gpu, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

    minVector<<<32, 32>>>(arr1_gpu, arr2_gpu, res_gpu, n);

    hipMemcpy(res, res_gpu, sizeof(double) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {
        cout << res[i] << " ";
    }

    cout << endl;

    return 0;
}
