
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

using namespace std;

__global__ void add(int a, int b, int* c) {
    *c = a + b;
}

int main() {
    // int n;
    // cin >> n;
    
    // double* arr1 = new double[n];
    // double* arr2 = new double[n];
    // double* res = new double[n];

    // for (int i = 0; i < n; ++i) {
    //     cin >> arr1[i];
    // }

    // for (int i = 0; i < n; ++i) {
    //     cin >> arr2[i];
    // }

    // for (int i = 0; i < n; ++i) {
    //     res[i] = max(arr1[i], arr2[i]);
    //     cout << res[i] << " ";
    // }

    int c;
    int* dev_c;

    hipMalloc((void**) &dev_c, sizeof(int));

    add<<<1, 1>>>(1, 2, dev_c);

    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    cout << c;

    return 0;
}
